
#include <hip/hip_runtime.h>
// ============================ MODES =========================================
// ============================================================================
#define EXEC_CPU 0
#define EXEC_OMP 1
#define EXEC_GPU 2

enum BoundaryCondition {
    Inflow,
    ZeroFlux,
};

enum Coordinates {
    Cartesian,
    Spherical,
};


// ============================ PHYSICS =======================================
// ============================================================================
#define NCONS 4
#define PLM_THETA 2.0
#define ADIABATIC_GAMMA (4.0 / 3.0)


// ============================ MATH ==========================================
// ============================================================================
#define min2(a, b) ((a) < (b) ? (a) : (b))
#define max2(a, b) ((a) > (b) ? (a) : (b))
#define min3(a, b, c) min2(a, min2(b, c))
#define max3(a, b, c) max2(a, max2(b, c))
#define sign(x) copysign(1.0, x)
#define minabs(a, b, c) min3(fabs(a), fabs(b), fabs(c))

static __device__ double plm_gradient_scalar(double yl, double y0, double yr)
{
    double a = (y0 - yl) * PLM_THETA;
    double b = (yr - yl) * 0.5;
    double c = (yr - y0) * PLM_THETA;
    return 0.25 * fabs(sign(a) + sign(b)) * (sign(a) + sign(c)) * minabs(a, b, c);
}

static __device__ void plm_gradient(double *yl, double *y0, double *yr, double *g)
{
    if (yl && y0 && yr)
    {
        for (int q = 0; q < NCONS; ++q)
        {
            g[q] = plm_gradient_scalar(yl[q], y0[q], yr[q]);
        }
    }
    else
    {
        for (int q = 0; q < NCONS; ++q)
        {
            g[q] = 0.0;
        }
    }
}


// ============================ HYDRO =========================================
// ============================================================================
static __device__ double primitive_to_gamma_beta_squared(const double *prim)
{
    const double u1 = prim[1];
    return u1 * u1;
}

static __device__ double primitive_to_lorentz_factor(const double *prim)
{
    return sqrt(1.0 + primitive_to_gamma_beta_squared(prim));
}

static __device__ double primitive_to_gamma_beta_component(const double *prim)
{
    return prim[1];
}

static __device__ double primitive_to_beta_component(const double *prim)
{
    const double w = primitive_to_lorentz_factor(prim);
    return prim[1] / w;
}

static __device__ double primitive_to_enthalpy_density(const double* prim)
{
    const double rho = prim[0];
    const double pre = prim[2];
    return rho + pre * (1.0 + 1.0 / (ADIABATIC_GAMMA - 1.0));
}

static __device__ void conserved_to_primitive(const double *cons, double *prim, double dv)
{
    const double newton_iter_max = 50;
    const double error_tolerance = 1e-12 * cons[0] / dv;
    const double gm              = ADIABATIC_GAMMA;
    const double m               = cons[0] / dv;
    const double tau             = cons[2] / dv;
    const double ss              = cons[1] / dv * cons[1] / dv;
    int iteration              = 0;
    double p                     = prim[2];
    double w0;

    while (1) {
        const double et = tau + p + m;
        const double b2 = min2(ss / et / et, 1.0 - 1e-10);
        const double w2 = 1.0 / (1.0 - b2);
        const double w  = sqrt(w2);
        const double e  = (tau + m * (1.0 - w) + p * (1.0 - w2)) / (m * w);
        const double d  = m / w;
        const double h  = 1.0 + e + p / d;
        const double a2 = gm * p / (d * h);
        const double f  = d * e * (gm - 1.0) - p;
        const double g  = b2 * a2 - 1.0;

        p -= f / g;

        if (fabs(f) < error_tolerance || iteration == newton_iter_max) {
            w0 = w;
            break;
        }
        iteration += 1;
    }

    prim[0] = m / w0;
    prim[1] = w0 * cons[1] / dv / (tau + m + p);
    prim[2] = p;
    prim[3] = cons[3] / cons[0];

    double mach_ceiling = 1000.0;
    double u = prim[1];
    double e = prim[2] / prim[0] * 3.0;
    double emin = u * u / (1.0 + u * u) / pow(mach_ceiling, 2.0);

    if (e < emin) {
        prim[2] = prim[0] * emin * (ADIABATIC_GAMMA - 1.0);
    }

    // if (prim[2] < 0.0 || prim[2] != prim[2]) {
    //     printf("[FATAL] srhd_1d got negative pressure p=%e at r=%e\n", prim[2], 0.0);
    //     exit(1);
    // }
}

static __device__ void primitive_to_conserved(const double *prim, double *cons, double dv)
{
    const double rho = prim[0];
    const double u1 = prim[1];
    const double pre = prim[2];

    const double w = primitive_to_lorentz_factor(prim);
    const double h = primitive_to_enthalpy_density(prim) / rho;
    const double m = rho * w;

    cons[0] = dv * m;
    cons[1] = dv * m * h * u1;
    cons[2] = dv * m * (h * w - 1.0) - dv * pre;
    cons[3] = dv * m * prim[3];
}

static __device__ void primitive_to_flux(const double *prim, const double *cons, double *flux)
{
    const double vn = primitive_to_beta_component(prim);
    const double pre = prim[2];
    const double s = prim[3]; // scalar concentration

    flux[0] = vn * cons[0];
    flux[1] = vn * cons[1] + pre;
    flux[2] = vn * cons[2] + pre * vn;
    flux[3] = vn * cons[0] * s;
}

static __device__ double primitive_to_sound_speed_squared(const double *prim)
{
    const double pre = prim[2];
    const double rho_h = primitive_to_enthalpy_density(prim);
    return ADIABATIC_GAMMA * pre / rho_h;
}

static __device__ void primitive_to_outer_wavespeeds(const double *prim, double *wavespeeds)
{
    const double a2 = primitive_to_sound_speed_squared(prim);
    const double un = primitive_to_gamma_beta_component(prim);
    const double uu = primitive_to_gamma_beta_squared(prim);
    const double vv = uu / (1.0 + uu);
    const double v2 = un * un / (1.0 + uu);
    const double vn = sqrt(v2);
    const double k0 = sqrt(a2 * (1.0 - vv) * (1.0 - vv * a2 - v2 * (1.0 - a2)));

    wavespeeds[0] = (vn * (1.0 - a2) - k0) / (1.0 - vv * a2);
    wavespeeds[1] = (vn * (1.0 - a2) + k0) / (1.0 - vv * a2);
}

static __device__ void riemann_hlle(const double *pl, const double *pr, double v_face, double *flux)
{
    double ul[NCONS];
    double ur[NCONS];
    double fl[NCONS];
    double fr[NCONS];
    double al[2];
    double ar[2];

    primitive_to_conserved(pl, ul, 1.0);
    primitive_to_conserved(pr, ur, 1.0);
    primitive_to_flux(pl, ul, fl);
    primitive_to_flux(pr, ur, fr);
    primitive_to_outer_wavespeeds(pl, al);
    primitive_to_outer_wavespeeds(pr, ar);

    const double am = min2(al[0], ar[0]);
    const double ap = max2(al[1], ar[1]);

    if (v_face < am)
    {
        for (int q = 0; q < NCONS; ++q)
        {
            flux[q] = fl[q] - v_face * ul[q];
        }
    }
    else if (v_face > ap)
    {
        for (int q = 0; q < NCONS; ++q)
        {
            flux[q] = fr[q] - v_face * ur[q];
        }
    }
    else
    {    
        for (int q = 0; q < NCONS; ++q)
        {
            double u_hll = (ur[q] * ap - ul[q] * am + (fl[q] - fr[q]))           / (ap - am);
            double f_hll = (fl[q] * ap - fr[q] * am - (ul[q] - ur[q]) * ap * am) / (ap - am);
            flux[q] = f_hll - v_face * u_hll;
        }
    }
}


// ============================ GEOMETRY ======================================
// ============================================================================
static __device__ double face_area(enum Coordinates coords, double x)
{
    switch (coords) {
        case Cartesian: return 1.0;
        case Spherical: return x * x;
    }
    return 0.0;
}

static __device__ double cell_volume(enum Coordinates coords, double x0, double x1) 
{
    switch (coords) {
        case Cartesian: return x1 - x0;
        case Spherical: return (pow(x1, 3.0) - pow(x0, 3.0)) / 3.0;
    }
    return 0.0;
}

static __device__ void geometric_source_terms(enum Coordinates coords, double x0, double x1, const double *prim, double *source)
{
    switch (coords) {
        case Spherical: {
            double p = prim[2];
            source[0] = 0.0;
            source[1] = p * (x1 * x1 - x0 * x0);
            source[2] = 0.0;
            source[3] = 0.0;
            break;
        }
        default: {
            source[0] = 0.0;
            source[1] = 0.0;
            source[2] = 0.0;
            source[3] = 0.0;
        }   
    }
}


// ============================ KERNELS =======================================
// ============================================================================


extern "C" void __global__ kernel_primitive_to_conserved(
    int num_zones,
    double *face_positions,
    double *primitive,
    double *conserved,
    double scale_factor,
    enum Coordinates coords)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= num_zones)
    {
        return;
    }
    double *p = &primitive[NCONS * i];
    double *u = &conserved[NCONS * i];
    double yl = face_positions[i];
    double yr = face_positions[i + 1];
    double xl = yl * scale_factor;
    double xr = yr * scale_factor;
    double dv = cell_volume(coords, xl, xr);
    primitive_to_conserved(p, u, dv);
}

extern "C" void __global__ kernel_conserved_to_primitive(
    int num_zones,
    double *face_positions,
    double *conserved,
    double *primitive,
    double scale_factor,
    enum Coordinates coords)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= num_zones)
    {
        return;
    }
    double *p = &primitive[NCONS * i];
    double *u = &conserved[NCONS * i];
    double yl = face_positions[i];
    double yr = face_positions[i + 1];
    double xl = yl * scale_factor;
    double xr = yr * scale_factor;
    double dv = cell_volume(coords, xl, xr);
    conserved_to_primitive(u, p, dv);
}


/**
 * Updates an array of primitive data by advancing it a single Runge-Kutta
 * step.
 *
 * @param num_zones              The number of zones in the array
 * @param face_positions[in]     [num_zones + 1] [1]
 * @param conserved_rk[in]       [num_zones] [NCONS]
 * @param primitive_rd[in]       [num_zones] [NCONS]
 * @param conserved_rd[in]       [num_zones] [NCONS]
 * @param conserved_wr[out]      [num_zones] [NCONS]
 * @param a0                     The scale factor at t=0
 * @param adot                   The expansion rate
 * @param a                      The RK averaging parameter
 * @param dt                     The time step
 * @param bc                     The boundary conditions type
 * @param coords                 The coordinate system
 */
extern "C" void __global__ kernel_advance_rk(
    int num_zones,
    double *face_positions,
    double *conserved_rk,
    double *primitive_rd,
    double *conserved_rd,
    double *conserved_wr,
    double a0,
    double adot,
    double t,
    double a,
    double dt,
    enum BoundaryCondition bc,
    enum Coordinates coords)
{
    #if (EXEC_MODE == EXEC_GPU)
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= num_zones) return;
    #elif (EXEC_MODE == EXEC_CPU)
    for (int i = 0; i < num_zones; ++i)
    #elif (EXEC_MODE == EXEC_OMP)
    #pragma omp parallel for
    for (int i = 0; i < num_zones; ++i)
    #endif

    {
        if (bc == Inflow && i == 0)
        {
            return;
        }

        int ni = num_zones;
        double yl = face_positions[i];
        double yr = face_positions[i + 1];
        double xl = yl * (a0 + adot * t);
        double xr = yr * (a0 + adot * t);

        double *urk = &conserved_rk[NCONS * i];
        double *prd = &primitive_rd[NCONS * i];
        double *urd = &conserved_rd[NCONS * i];
        double *uwr = &conserved_wr[NCONS * i];
        double *pli = i >= 0 + 1 ? &primitive_rd[NCONS * (i - 1)] : NULL;
        double *pri = i < ni - 1 ? &primitive_rd[NCONS * (i + 1)] : NULL;
        double *pki = i >= 0 + 2 ? &primitive_rd[NCONS * (i - 2)] : NULL;
        double *pti = i < ni - 2 ? &primitive_rd[NCONS * (i + 2)] : NULL;

        double plip[NCONS];
        double plim[NCONS];
        double prip[NCONS];
        double prim[NCONS];
        double gxli[NCONS];
        double gxri[NCONS];
        double gxcc[NCONS];

        // NOTE: the gradient calculation here assumes smoothly varying face
        // separations. Also note plm_gradient initializes the gradients to zero
        // if any of the inputs are NULL.
        plm_gradient(pki, pli, prd, gxli);
        plm_gradient(pli, prd, pri, gxcc);
        plm_gradient(prd, pri, pti, gxri);

        for (int q = 0; q < NCONS; ++q)
        {
            plim[q] = pli ? pli[q] + 0.5 * gxli[q] : prd[q];
            plip[q] = prd[q] - 0.5 * gxcc[q];
            prim[q] = prd[q] + 0.5 * gxcc[q];
            prip[q] = pri ? pri[q] - 0.5 * gxri[q] : prd[q];
        }

        double fli[NCONS];
        double fri[NCONS];
        double sources[NCONS];
        double dal = face_area(coords, xl);
        double dar = face_area(coords, xr);

        riemann_hlle(plim, plip, yl * adot, fli);
        riemann_hlle(prim, prip, yr * adot, fri);
        geometric_source_terms(coords, xl, xr, prd, sources);

        for (int q = 0; q < NCONS; ++q)
        {
            uwr[q] = urd[q] + (fli[q] * dal - fri[q] * dar + sources[q]) * dt;
            uwr[q] = (1.0 - a) * uwr[q] + a * urk[q];
        }
    }
}
